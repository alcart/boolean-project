
#include <hip/hip_runtime.h>
__device__ char getBits(long i, int idx) {
    return (i & (1 << idx)) != 0;
}

__global__ void gpu_boolean_matcher(char* result_ptr, int index){ 

char result = *result_ptr;
long element = (((long)blockIdx.x)*((long)blockDim.x) + ((long)threadIdx.x)); 
long maxCores = gridDim.x; 

switch(index){

//Expression #1, Amount of variables: 2
case 1:
    for (long i = element; i < (((long) 1) << 2); i += maxCores) {
        result = result && (getBits(i, 0)==getBits(i, 0)&&getBits(i, 1));
    }
    break;
// Expression #2, Amount of variables: 2
case 2:
    for (long i = element; i < (((long) 1) << 2); i += maxCores) {
        result = result && (getBits(i, 0)==!getBits(i, 0)||getBits(i, 1));
    }
    break;
// Expression #3, Amount of variables: 2
case 3:
    for (long i = element; i < (((long) 1) << 2); i += maxCores) {
        result = result && (getBits(i, 0)==!getBits(i, 1)&&!getBits(i, 0));
    }
    break;
// Expression #4, Amount of variables: 3
case 4:
    for (long i = element; i < (((long) 1) << 3); i += maxCores) {
        result = result && (getBits(i, 0)==(getBits(i, 1)&&!getBits(i, 0))&&getBits(i, 2));
    }
    break;
// Expression #5, Amount of variables: 4
case 5:
    for (long i = element; i < (((long) 1) << 4); i += maxCores) {
        result = result && (getBits(i, 0)==(!getBits(i, 1)||getBits(i, 0))&&(getBits(i, 2)||getBits(i, 3)));
    }
    break;
// Expression #6, Amount of variables: 5
case 6:
    for (long i = element; i < (((long) 1) << 5); i += maxCores) {
        result = result && (getBits(i, 0)==(getBits(i, 1)&&!getBits(i, 2))&&(getBits(i, 3)&&(getBits(i, 4)||!getBits(i, 0))));
    }
    break;
// Expression #7, Amount of variables: 20
case 7:
    for (long i = element; i < (((long) 1) << 20); i += maxCores) {
        result = result && (getBits(i, 0)==(getBits(i, 0)||!getBits(i, 1))&&(!getBits(i, 2)||getBits(i, 3))&&(((!getBits(i, 4)||!getBits(i, 5))||(!getBits(i, 6)||getBits(i, 7))))&&(!getBits(i, 8)||getBits(i, 9))&&(!getBits(i, 10)||!getBits(i, 11))&&(((!getBits(i, 12)||!getBits(i, 13))||getBits(i, 14))&&(getBits(i, 15)||getBits(i, 16))&&(getBits(i, 17)&&(!getBits(i, 18)&&!getBits(i, 19)))));
    }
    break;
// Expression #8, Amount of variables: 21
case 8:
    for (long i = element; i < (((long) 1) << 21); i += maxCores) {
        result = result && (getBits(i, 0)==((!(!getBits(i, 0)||(getBits(i, 1)||!getBits(i, 2))))&&((getBits(i, 3)&&!getBits(i, 4))&&(getBits(i, 5)||!getBits(i, 6))))||(((!getBits(i, 7)||!getBits(i, 8))||!getBits(i, 9))&&((getBits(i, 10)&&!getBits(i, 11))&&(getBits(i, 12)||!getBits(i, 13))))||((!(getBits(i, 14)||(getBits(i, 15)||getBits(i, 16))))&&((getBits(i, 17)||!getBits(i, 18))&&(getBits(i, 19)||getBits(i, 20)))));
    }
    break;
// Expression #9, Amount of variables: 25
case 9:
    for (long i = element; i < (((long) 1) << 25); i += maxCores) {
        result = result && (getBits(i, 0)==((getBits(i, 1)||getBits(i, 0))||(getBits(i, 2)&&(getBits(i, 3)&&getBits(i, 4))))||((getBits(i, 5)||getBits(i, 6))||((!getBits(i, 7)&&getBits(i, 8))&&!getBits(i, 9)))||((getBits(i, 10)&&!getBits(i, 11))||(!getBits(i, 12)&&(!getBits(i, 13)&&!getBits(i, 14))))||((getBits(i, 15)||(getBits(i, 16)||getBits(i, 17)))||(!getBits(i, 18)&&!getBits(i, 19)))||((!getBits(i, 20)||!getBits(i, 21))||((!getBits(i, 22)||!getBits(i, 23))||!getBits(i, 24))));
    }
    break;
// Expression #10, Amount of variables: 21
case 10:
    for (long i = element; i < (((long) 1) << 21); i += maxCores) {
        result = result && (getBits(i, 0)==(((!getBits(i, 1)&&getBits(i, 0))&&!getBits(i, 2))&&((getBits(i, 3)||!getBits(i, 4))&&(getBits(i, 5)||!getBits(i, 6))))||(((!getBits(i, 7)&&getBits(i, 8))||!getBits(i, 9))&&((getBits(i, 10)||!getBits(i, 11))&&(getBits(i, 12)&&!getBits(i, 13))))||(((getBits(i, 14)&&getBits(i, 15))||getBits(i, 16))&&((getBits(i, 17)||!getBits(i, 18))&&(!getBits(i, 19)&&getBits(i, 20)))));
    }
    break;
// Expression #11, Amount of variables: 30
case 11:
    for (long i = element; i < (((long) 1) << 30); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)||getBits(i, 2)))||(!getBits(i, 3)&&(!getBits(i, 4)&&!getBits(i, 0))))||((!(getBits(i, 5)||getBits(i, 6)))||(getBits(i, 7)&&(getBits(i, 8)&&getBits(i, 9))))||((!(!getBits(i, 10)||getBits(i, 11)))||((getBits(i, 12)||getBits(i, 13))&&getBits(i, 14)))||((!(getBits(i, 15)||getBits(i, 16)))||((!getBits(i, 17)&&!getBits(i, 18))&&!getBits(i, 19)))||((!(getBits(i, 20)||getBits(i, 21)))||((getBits(i, 22)&&getBits(i, 23))&&getBits(i, 24)))||((!(!getBits(i, 25)||getBits(i, 26)))||((getBits(i, 27)||getBits(i, 28))&&getBits(i, 29))));
    }
    break;
// Expression #12, Amount of variables: 21
case 12:
    for (long i = element; i < (((long) 1) << 21); i += maxCores) {
        result = result && (getBits(i, 0)==((!getBits(i, 1)&&(getBits(i, 2)&&!getBits(i, 0)))||((getBits(i, 3)||!getBits(i, 4))&&(getBits(i, 5)||!getBits(i, 6))))&&(((!getBits(i, 7)&&getBits(i, 8))||!getBits(i, 9))&&((getBits(i, 10)||!getBits(i, 11))&&(getBits(i, 12)&&!getBits(i, 13))))||(((getBits(i, 14)||getBits(i, 15))&&getBits(i, 16))&&((getBits(i, 17)||!getBits(i, 18))&&!getBits(i, 19))&&getBits(i, 20)));
    }
    break;
// Expression #13, Amount of variables: 39
case 13:
    for (long i = element; i < (((long) 1) << 39); i += maxCores) {
        result = result && (getBits(i, 0)==((!getBits(i, 1)||getBits(i, 2))||(!getBits(i, 3)&&(!getBits(i, 4)||!getBits(i, 5))))||((!(getBits(i, 0)||getBits(i, 6)))||(getBits(i, 7)||(getBits(i, 8)&&getBits(i, 9))))||((!(!getBits(i, 10)||getBits(i, 11)))||((getBits(i, 12)||getBits(i, 13))&&getBits(i, 14)))||((!(getBits(i, 15)&&getBits(i, 16)))||((!getBits(i, 17)&&!getBits(i, 18))||!getBits(i, 19)))||((!(getBits(i, 20)||getBits(i, 21)))||((getBits(i, 22)||getBits(i, 23))&&getBits(i, 24)))||((!(!getBits(i, 25)||getBits(i, 26)))||((getBits(i, 27)||getBits(i, 28))&&getBits(i, 29)))&&((!(getBits(i, 30)||getBits(i, 21)))||((getBits(i, 31)||getBits(i, 32))&&getBits(i, 33)))||((!(!getBits(i, 34)||getBits(i, 35)))||((getBits(i, 36)||getBits(i, 37))&&getBits(i, 38))));
    }
    break;
// Expression #14, Amount of variables: 45
case 14:
    for (long i = element; i < (((long) 1) << 45); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)||getBits(i, 0)))||(!getBits(i, 2)&&(!getBits(i, 3)&&(!getBits(i, 4)||getBits(i, 5)))))||((!(getBits(i, 6)||(getBits(i, 7)&&getBits(i, 8))))||(getBits(i, 9)||(getBits(i, 10)&&getBits(i, 11))))||((!(!getBits(i, 12)||getBits(i, 13)))||((getBits(i, 14)||getBits(i, 15))&&(getBits(i, 16)||getBits(i, 17))))||((!(getBits(i, 18)||getBits(i, 19)))||((!getBits(i, 20)&&!getBits(i, 21))||!getBits(i, 22)))||((!(getBits(i, 23)&&(getBits(i, 24)||getBits(i, 25))))||((getBits(i, 26)||getBits(i, 27))&&getBits(i, 28)))||((!(!getBits(i, 29)||getBits(i, 30)))||((getBits(i, 31)||getBits(i, 32))&&(getBits(i, 33)||getBits(i, 34))))||((!(getBits(i, 35)||getBits(i, 36)))||((getBits(i, 37)&&getBits(i, 38))&&getBits(i, 39)))&&((getBits(i, 40)||!getBits(i, 41))&&(!getBits(i, 42)||(getBits(i, 43)&&getBits(i, 44)))));
    }
    break;
// Expression #15, Amount of variables: 50
case 15:
    for (long i = element; i < (((long) 1) << 50); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)||getBits(i, 0)))&&(!getBits(i, 2)&&(!getBits(i, 3)&&(!getBits(i, 4)||getBits(i, 5)))))||((!(getBits(i, 6)||(getBits(i, 7)&&getBits(i, 8))))||(getBits(i, 9)||(getBits(i, 10)&&getBits(i, 11))))||((!(!getBits(i, 12)||getBits(i, 13)))||((getBits(i, 14)||getBits(i, 15))&&(getBits(i, 16)||getBits(i, 17))))&&((!(getBits(i, 18)||getBits(i, 19)))||((!getBits(i, 20)&&!getBits(i, 21))||!getBits(i, 22)))||((!(getBits(i, 23)&&(getBits(i, 24)||getBits(i, 25))))||((getBits(i, 26)||getBits(i, 27))&&getBits(i, 28)))||((!(!getBits(i, 29)||getBits(i, 30)))||((getBits(i, 31)||getBits(i, 32))&&(getBits(i, 33)||getBits(i, 34))))||((!(getBits(i, 35)||getBits(i, 36)))||((getBits(i, 37)&&getBits(i, 38))&&getBits(i, 39)))&&((getBits(i, 40)||!getBits(i, 41))&&(!getBits(i, 42)||(getBits(i, 43)&&getBits(i, 44))))||(getBits(i, 45)&&(getBits(i, 46)||(!getBits(i, 47)&&(!getBits(i, 48)||getBits(i, 49))))));
    }
    break;
// Expression #16, Amount of variables: 60
case 16:
    for (long i = element; i < (((long) 1) << 60); i += maxCores) {
        result = result && (getBits(i, 0)==((getBits(i, 0)&&getBits(i, 1))||((!getBits(i, 2)||!getBits(i, 3))&&!getBits(i, 4)))&&((getBits(i, 5)&&getBits(i, 6))||((!getBits(i, 7)&&!getBits(i, 8))||!getBits(i, 9)))&&((getBits(i, 10)&&getBits(i, 11))||((!getBits(i, 12)||!getBits(i, 13))&&!getBits(i, 14)))&&((getBits(i, 15)&&getBits(i, 16))||(!getBits(i, 17)&&(!getBits(i, 18)||!getBits(i, 19))))&&((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||(!getBits(i, 27)||(getBits(i, 28)&&getBits(i, 29))))&&((getBits(i, 30)&&getBits(i, 31))||((!getBits(i, 32)||getBits(i, 33))&&!getBits(i, 34)))&&((getBits(i, 35)&&getBits(i, 36))||(getBits(i, 37)&&(getBits(i, 38)||!getBits(i, 39))))&&((getBits(i, 40)&&getBits(i, 41))||(!getBits(i, 42)&&!getBits(i, 43)&&!getBits(i, 44)))&&((getBits(i, 45)&&getBits(i, 46))||((!getBits(i, 47)||getBits(i, 48))&&!getBits(i, 49)))&&((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)&&!getBits(i, 53)&&!getBits(i, 54)))&&((getBits(i, 55)&&getBits(i, 56))||(!getBits(i, 57)&&(!getBits(i, 58)||!getBits(i, 59)))));
    }
    break;
// Expression #17, Amount of variables: 61
case 17:
    for (long i = element; i < (((long) 1) << 61); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 0)&&getBits(i, 1)))||((!getBits(i, 2)||!getBits(i, 3))&&!getBits(i, 4)))||((!getBits(i, 5)&&getBits(i, 6))||(!getBits(i, 7)&&(!getBits(i, 8)||!getBits(i, 9))))&&((getBits(i, 10)&&getBits(i, 11))||(!getBits(i, 12)&&!getBits(i, 13)&&!getBits(i, 14)))&&((getBits(i, 15)&&getBits(i, 16))&&((!(getBits(i, 17)&&!getBits(i, 18)))&&!getBits(i, 19)))||((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||((!getBits(i, 27)||!getBits(i, 28))&&!getBits(i, 29)))&&((getBits(i, 30)&&getBits(i, 31))||(!getBits(i, 32)&&!getBits(i, 33)&&!getBits(i, 34)))||((getBits(i, 35)&&!getBits(i, 36))||(getBits(i, 37)&&(getBits(i, 38)||getBits(i, 39))))&&((getBits(i, 40)&&getBits(i, 41))||(getBits(i, 42)&&(getBits(i, 43)&&!getBits(i, 44))))&&((!(getBits(i, 45)&&getBits(i, 46)))||((getBits(i, 47)&&getBits(i, 48))||!getBits(i, 49)))||((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)&&(getBits(i, 53)||!getBits(i, 54))))&&((getBits(i, 55)&&getBits(i, 56))||((getBits(i, 57)&&getBits(i, 58))||(getBits(i, 59)&&!getBits(i, 60)))));
    }
    break;
// Expression #18, Amount of variables: 60
case 18:
    for (long i = element; i < (((long) 1) << 60); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)&&getBits(i, 0)))||((!getBits(i, 2)||!getBits(i, 3))&&!getBits(i, 4)))||((!getBits(i, 5)&&getBits(i, 6))||(!getBits(i, 7)||(!getBits(i, 8)&&!getBits(i, 9))))&&((getBits(i, 10)&&getBits(i, 11))||(!getBits(i, 12)&&!getBits(i, 13)&&!getBits(i, 14)))&&((getBits(i, 15)||getBits(i, 16))&&((!(getBits(i, 17)&&!getBits(i, 18)))&&!getBits(i, 19)))||((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||((!getBits(i, 27)&&!getBits(i, 28))||!getBits(i, 29)))&&((getBits(i, 30)&&getBits(i, 31))||((!getBits(i, 32)&&!getBits(i, 33))&&!getBits(i, 34)))||((getBits(i, 35)&&!getBits(i, 36))||(getBits(i, 37)&&getBits(i, 38)&&getBits(i, 39)))&&((getBits(i, 40)&&getBits(i, 41))||(getBits(i, 42)&&(getBits(i, 43)&&!getBits(i, 44))))&&((!(getBits(i, 45)&&getBits(i, 46)))||((getBits(i, 47)&&getBits(i, 48))||!getBits(i, 49)))||((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)&&getBits(i, 53)&&!getBits(i, 54)))&&((getBits(i, 55)&&getBits(i, 56))||((getBits(i, 57)&&getBits(i, 58))||getBits(i, 59))));
    }
    break;
// Expression #19, Amount of variables: 62
case 19:
    for (long i = element; i < (((long) 1) << 62); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)&&getBits(i, 0)))||(!getBits(i, 2)&&!getBits(i, 3)&&!getBits(i, 4)))||((!getBits(i, 5)&&getBits(i, 6))||(!getBits(i, 7)||(!getBits(i, 8)&&!getBits(i, 9))))&&((getBits(i, 10)&&getBits(i, 11))||((!getBits(i, 12)&&!getBits(i, 13))||!getBits(i, 14)))&&((getBits(i, 15)&&getBits(i, 16))&&((!(getBits(i, 17)&&!getBits(i, 18)))||!getBits(i, 19)))||((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||((!getBits(i, 27)&&!getBits(i, 28))||!getBits(i, 29)))&&((getBits(i, 30)&&getBits(i, 31))||((!getBits(i, 32)||!getBits(i, 33))&&!getBits(i, 34)))||((getBits(i, 35)&&!getBits(i, 36))||(getBits(i, 37)||(getBits(i, 38)&&getBits(i, 39))))&&((getBits(i, 40)&&getBits(i, 41))||(getBits(i, 42)||(getBits(i, 43)&&!getBits(i, 44))))&&((!(getBits(i, 45)&&getBits(i, 46)))||((getBits(i, 47)&&getBits(i, 48))||!getBits(i, 49)))||((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)&&(getBits(i, 53)||!getBits(i, 54))))&&((getBits(i, 55)&&getBits(i, 56))||((getBits(i, 57)&&getBits(i, 58))||getBits(i, 59)))||(getBits(i, 60)&&(!getBits(i, 61))));
    }
    break;
// Expression #20, Amount of variables: 60
case 20:
    for (long i = element; i < (((long) 1) << 60); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)||getBits(i, 2)))||((!getBits(i, 3)||!getBits(i, 4))&&!getBits(i, 0)))||((!getBits(i, 5)&&getBits(i, 6))||(!getBits(i, 7)||(!getBits(i, 8)&&!getBits(i, 9))))&&((getBits(i, 10)&&getBits(i, 11))||((!getBits(i, 12)&&!getBits(i, 13))||!getBits(i, 14)))&&((getBits(i, 15)||getBits(i, 16))&&((!(getBits(i, 17)&&!getBits(i, 18)))&&!getBits(i, 19)))||((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||(!getBits(i, 27)&&!getBits(i, 28)))&&(((getBits(i, 29)||(getBits(i, 30)&&getBits(i, 31)))||((!getBits(i, 32)||!getBits(i, 33))&&!getBits(i, 34))))||((getBits(i, 35)&&!getBits(i, 36))||(getBits(i, 37)&&(getBits(i, 38)||getBits(i, 39))))&&((getBits(i, 40)&&getBits(i, 41))||(getBits(i, 42)||(getBits(i, 43)&&!getBits(i, 44))))&&((!(getBits(i, 45)&&getBits(i, 46)))||((getBits(i, 47)&&getBits(i, 48))||!getBits(i, 49)))||((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)||(getBits(i, 53)&&!getBits(i, 54))))&&((getBits(i, 55)&&getBits(i, 56))||((getBits(i, 57)&&getBits(i, 58))||getBits(i, 59))));
    }
    break;
// Expression #21, Amount of variables: 63
case 21:
    for (long i = element; i < (((long) 1) << 63); i += maxCores) {
        result = result && (getBits(i, 0)==((!(getBits(i, 1)&&getBits(i, 2)))||((!getBits(i, 3)||!getBits(i, 4))&&!getBits(i, 0)))||((!getBits(i, 5)&&getBits(i, 6))||(!getBits(i, 7)&&(!getBits(i, 8)||!getBits(i, 9))))&&((getBits(i, 10)&&getBits(i, 11))||((!getBits(i, 12)||!getBits(i, 13))&&!getBits(i, 14)))&&((getBits(i, 15)&&getBits(i, 16))&&((!(getBits(i, 17)&&!getBits(i, 18)))&&!getBits(i, 19)))||((getBits(i, 20)&&getBits(i, 21))||(!getBits(i, 22)||(!getBits(i, 23)&&!getBits(i, 24))))&&((getBits(i, 25)&&getBits(i, 26))||((!getBits(i, 27)||!getBits(i, 28))&&!getBits(i, 29)))&&((getBits(i, 30)&&getBits(i, 31))||((!getBits(i, 32)&&!getBits(i, 33))||!getBits(i, 34)))||((getBits(i, 35)&&!getBits(i, 36))||(getBits(i, 37)&&(getBits(i, 38)||getBits(i, 39))))&&((getBits(i, 40)&&getBits(i, 41))||(getBits(i, 42)&&(getBits(i, 43)||!getBits(i, 44))))&&((!(getBits(i, 45)&&getBits(i, 46)))||((getBits(i, 47)&&!getBits(i, 48))||!getBits(i, 49)))||((getBits(i, 50)&&getBits(i, 51))||(!getBits(i, 52)||(getBits(i, 53)&&!getBits(i, 54))))&&((getBits(i, 55)&&getBits(i, 56))||((getBits(i, 57)&&getBits(i, 58))||getBits(i, 59)))||((getBits(i, 60)&&!getBits(i, 61))||(!getBits(i, 62))));
    }
    break;
// Expression #22, Amount of variables: 2
case 22:
    for (long i = element; i < (((long) 1) << 2); i += maxCores) {
        result = result && (getBits(i, 0)||!getBits(i, 1));
    }
    break;
// Expression #23, Amount of variables: 4
case 23:
    for (long i = element; i < (((long) 1) << 4); i += maxCores) {
        result = result && (getBits(i, 0)&&!getBits(i, 1)||getBits(i, 2)||getBits(i, 3));
    }
    break;
// Expression #24, Amount of variables: 5
case 24:
    for (long i = element; i < (((long) 1) << 5); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)&&!getBits(i, 1)||getBits(i, 3)||getBits(i, 4));
    }
    break;
// Expression #25, Amount of variables: 8
case 25:
    for (long i = element; i < (((long) 1) << 8); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)^!getBits(i, 7));
    }
    break;
// Expression #26, Amount of variables: 9
case 26:
    for (long i = element; i < (((long) 1) << 9); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)||getBits(i, 1)||getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||!getBits(i, 8));
    }
    break;
// Expression #27, Amount of variables: 12
case 27:
    for (long i = element; i < (((long) 1) << 12); i += maxCores) {
        result = result && (getBits(i, 0)||!getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)&&!getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 10)&&getBits(i, 11));
    }
    break;
// Expression #28, Amount of variables: 10
case 28:
    for (long i = element; i < (((long) 1) << 10); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)^!getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)||!getBits(i, 5)||getBits(i, 9)^getBits(i, 5)||getBits(i, 5)||getBits(i, 8));
    }
    break;
// Expression #29, Amount of variables: 12
case 29:
    for (long i = element; i < (((long) 1) << 12); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)^!getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)^!getBits(i, 7)||getBits(i, 3)^getBits(i, 0)||getBits(i, 8)&&getBits(i, 9)||getBits(i, 0)||getBits(i, 10)||getBits(i, 11)&&getBits(i, 10));
    }
    break;
// Expression #30, Amount of variables: 16
case 30:
    for (long i = element; i < (((long) 1) << 16); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)&&getBits(i, 3)||getBits(i, 5)&&getBits(i, 6)||getBits(i, 7)^getBits(i, 8)||getBits(i, 3)&&getBits(i, 9)||getBits(i, 10)||!getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)||getBits(i, 15));
    }
    break;
// Expression #31, Amount of variables: 18
case 31:
    for (long i = element; i < (((long) 1) << 18); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)&&getBits(i, 9)||getBits(i, 10)^getBits(i, 11)||getBits(i, 12)^getBits(i, 13)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)&&!getBits(i, 0)||getBits(i, 16)||!getBits(i, 17));
    }
    break;
// Expression #32, Amount of variables: 18
case 32:
    for (long i = element; i < (((long) 1) << 18); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||!getBits(i, 5)||getBits(i, 6)^getBits(i, 7)||getBits(i, 8)^!getBits(i, 9)||getBits(i, 10)&&!getBits(i, 11)||getBits(i, 4)^getBits(i, 12)||getBits(i, 13)&&getBits(i, 14)||getBits(i, 11)^!getBits(i, 10)||getBits(i, 15)^getBits(i, 16)||getBits(i, 17)&&getBits(i, 7));
    }
    break;
// Expression #33, Amount of variables: 21
case 33:
    for (long i = element; i < (((long) 1) << 21); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)&&getBits(i, 0)||getBits(i, 7)&&getBits(i, 8)||getBits(i, 9)||getBits(i, 10)||getBits(i, 11)&&getBits(i, 12)||getBits(i, 13)||getBits(i, 14)||getBits(i, 15)||getBits(i, 16)||getBits(i, 17)&&getBits(i, 18)||getBits(i, 19)^getBits(i, 2)||getBits(i, 5)^getBits(i, 20));
    }
    break;
// Expression #34, Amount of variables: 24
case 34:
    for (long i = element; i < (((long) 1) << 24); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)^!getBits(i, 11)||getBits(i, 12)&&getBits(i, 13)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)&&getBits(i, 17)||getBits(i, 18)||getBits(i, 5)||getBits(i, 19)||getBits(i, 20)||getBits(i, 21)^!getBits(i, 22)||getBits(i, 23)^getBits(i, 15));
    }
    break;
// Expression #35, Amount of variables: 26
case 35:
    for (long i = element; i < (((long) 1) << 26); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)&&!getBits(i, 9)||getBits(i, 10)||getBits(i, 11)||getBits(i, 12)^!getBits(i, 13)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)||getBits(i, 8)||getBits(i, 17)&&getBits(i, 18)||getBits(i, 19)||getBits(i, 20)||getBits(i, 15)^!getBits(i, 21)||getBits(i, 22)||!getBits(i, 23)||getBits(i, 24)&&getBits(i, 25));
    }
    break;
// Expression #36, Amount of variables: 28
case 36:
    for (long i = element; i < (((long) 1) << 28); i += maxCores) {
        result = result && (getBits(i, 0)^!getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)^getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 10)||!getBits(i, 11)||getBits(i, 3)^getBits(i, 12)||getBits(i, 13)&&!getBits(i, 14)||getBits(i, 15)^getBits(i, 16)||getBits(i, 17)^getBits(i, 12)||getBits(i, 18)||getBits(i, 19)||getBits(i, 20)^!getBits(i, 21)||getBits(i, 22)^getBits(i, 23)||getBits(i, 24)||getBits(i, 25)||getBits(i, 26)^getBits(i, 27));
    }
    break;
// Expression #37, Amount of variables: 29
case 37:
    for (long i = element; i < (((long) 1) << 29); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)||!getBits(i, 7)||getBits(i, 1)&&getBits(i, 8)||getBits(i, 9)||!getBits(i, 10)||getBits(i, 11)^!getBits(i, 12)||getBits(i, 13)&&getBits(i, 14)||getBits(i, 12)&&!getBits(i, 15)||getBits(i, 16)&&!getBits(i, 17)||getBits(i, 18)^getBits(i, 19)||getBits(i, 20)^getBits(i, 21)||getBits(i, 22)&&getBits(i, 23)||getBits(i, 24)||getBits(i, 25)||getBits(i, 26)&&getBits(i, 27)||getBits(i, 15)^getBits(i, 28));
    }
    break;
// Expression #38, Amount of variables: 27
case 38:
    for (long i = element; i < (((long) 1) << 27); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||!getBits(i, 5)||getBits(i, 6)^!getBits(i, 7)||getBits(i, 8)^!getBits(i, 9)||getBits(i, 10)&&!getBits(i, 11)||getBits(i, 1)^getBits(i, 12)||getBits(i, 13)&&getBits(i, 14)||getBits(i, 14)^getBits(i, 15)||getBits(i, 16)||!getBits(i, 6)||getBits(i, 4)^getBits(i, 17)||getBits(i, 18)||!getBits(i, 19)||getBits(i, 7)||!getBits(i, 20)||getBits(i, 21)&&getBits(i, 9)||getBits(i, 22)^getBits(i, 23)||getBits(i, 9)&&getBits(i, 24)||getBits(i, 25)^!getBits(i, 26));
    }
    break;
// Expression #39, Amount of variables: 29
case 39:
    for (long i = element; i < (((long) 1) << 29); i += maxCores) {
        result = result && (getBits(i, 0)&&!getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)^!getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)&&!getBits(i, 0)||getBits(i, 9)||getBits(i, 10)||getBits(i, 11)&&getBits(i, 12)||getBits(i, 13)&&getBits(i, 2)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)^getBits(i, 5)||getBits(i, 17)&&getBits(i, 18)||getBits(i, 19)||getBits(i, 20)||getBits(i, 21)||!getBits(i, 18)||getBits(i, 8)^getBits(i, 22)||getBits(i, 23)&&getBits(i, 3)||getBits(i, 24)^getBits(i, 25)||getBits(i, 26)||!getBits(i, 27)||getBits(i, 28)^!getBits(i, 4));
    }
    break;
// Expression #40, Amount of variables: 31
case 40:
    for (long i = element; i < (((long) 1) << 31); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)^getBits(i, 7)||getBits(i, 8)^getBits(i, 3)||getBits(i, 9)&&!getBits(i, 10)||getBits(i, 11)||getBits(i, 12)||getBits(i, 13)&&!getBits(i, 14)||getBits(i, 15)||getBits(i, 16)||getBits(i, 1)||getBits(i, 7)||getBits(i, 14)^!getBits(i, 7)||getBits(i, 17)^getBits(i, 18)||getBits(i, 19)^getBits(i, 20)||getBits(i, 21)^getBits(i, 22)||getBits(i, 9)||!getBits(i, 23)||getBits(i, 24)&&!getBits(i, 25)||getBits(i, 26)||!getBits(i, 27)||getBits(i, 28)||getBits(i, 12)||getBits(i, 29)^!getBits(i, 30));
    }
    break;
// Expression #41, Amount of variables: 34
case 41:
    for (long i = element; i < (((long) 1) << 34); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||!getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 4)&&getBits(i, 8)||getBits(i, 9)||getBits(i, 10)||getBits(i, 11)^getBits(i, 12)||getBits(i, 13)^getBits(i, 14)||getBits(i, 15)^!getBits(i, 15)||getBits(i, 16)&&!getBits(i, 17)||getBits(i, 18)&&getBits(i, 19)||getBits(i, 20)^getBits(i, 21)||getBits(i, 22)^getBits(i, 23)||getBits(i, 14)||getBits(i, 24)||getBits(i, 19)^getBits(i, 25)||getBits(i, 26)^!getBits(i, 27)||getBits(i, 28)&&!getBits(i, 29)||getBits(i, 5)^getBits(i, 30)||getBits(i, 31)||!getBits(i, 32)||getBits(i, 31)||getBits(i, 33));
    }
    break;
// Expression #42, Amount of variables: 35
case 42:
    for (long i = element; i < (((long) 1) << 35); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)^!getBits(i, 5)||getBits(i, 6)^!getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)&&getBits(i, 11)||getBits(i, 12)^!getBits(i, 13)||getBits(i, 14)&&getBits(i, 11)||getBits(i, 15)&&getBits(i, 13)||getBits(i, 16)||getBits(i, 17)||getBits(i, 18)&&getBits(i, 19)||getBits(i, 20)^getBits(i, 21)||getBits(i, 22)&&getBits(i, 23)||getBits(i, 24)^!getBits(i, 1)||getBits(i, 25)||getBits(i, 9)||getBits(i, 26)&&getBits(i, 27)||getBits(i, 28)||!getBits(i, 29)||getBits(i, 30)&&getBits(i, 31)||getBits(i, 13)&&getBits(i, 7)||getBits(i, 8)||!getBits(i, 32)||getBits(i, 33)&&getBits(i, 34));
    }
    break;
// Expression #43, Amount of variables: 36
case 43:
    for (long i = element; i < (((long) 1) << 36); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)^!getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)&&getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)&&!getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)&&getBits(i, 0)||getBits(i, 19)&&getBits(i, 20)||getBits(i, 21)^getBits(i, 22)||getBits(i, 23)^getBits(i, 0)||getBits(i, 24)||!getBits(i, 25)||getBits(i, 22)&&getBits(i, 26)||getBits(i, 14)^getBits(i, 27)||getBits(i, 28)^getBits(i, 9)||getBits(i, 29)^getBits(i, 30)||getBits(i, 31)||!getBits(i, 32)||getBits(i, 33)&&!getBits(i, 1)||getBits(i, 5)||getBits(i, 17)||getBits(i, 34)&&getBits(i, 35));
    }
    break;
// Expression #44, Amount of variables: 40
case 44:
    for (long i = element; i < (((long) 1) << 40); i += maxCores) {
        result = result && (getBits(i, 0)||!getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 6)||!getBits(i, 7)||getBits(i, 8)^getBits(i, 2)||getBits(i, 2)||getBits(i, 9)||getBits(i, 10)^getBits(i, 11)||getBits(i, 12)||getBits(i, 11)||getBits(i, 13)&&getBits(i, 14)||getBits(i, 15)&&getBits(i, 16)||getBits(i, 17)||!getBits(i, 18)||getBits(i, 19)&&getBits(i, 20)||getBits(i, 21)^getBits(i, 22)||getBits(i, 23)&&getBits(i, 20)||getBits(i, 24)||getBits(i, 25)||getBits(i, 26)^getBits(i, 27)||getBits(i, 28)||!getBits(i, 14)||getBits(i, 29)||getBits(i, 30)||getBits(i, 31)||getBits(i, 32)||getBits(i, 33)^!getBits(i, 34)||getBits(i, 35)^getBits(i, 36)||getBits(i, 37)&&!getBits(i, 38)||getBits(i, 4)||getBits(i, 39));
    }
    break;
// Expression #45, Amount of variables: 39
case 45:
    for (long i = element; i < (((long) 1) << 39); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 10)^!getBits(i, 11)||getBits(i, 12)^!getBits(i, 13)||getBits(i, 14)^!getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)||getBits(i, 19)||getBits(i, 20)&&getBits(i, 21)||getBits(i, 22)||getBits(i, 23)||getBits(i, 24)||getBits(i, 7)||getBits(i, 25)||!getBits(i, 26)||getBits(i, 27)||getBits(i, 28)||getBits(i, 12)||getBits(i, 29)||getBits(i, 19)^!getBits(i, 30)||getBits(i, 31)^!getBits(i, 1)||getBits(i, 32)&&!getBits(i, 15)||getBits(i, 33)||!getBits(i, 34)||getBits(i, 35)||getBits(i, 1)||getBits(i, 36)||getBits(i, 37)||getBits(i, 37)||getBits(i, 5)||getBits(i, 11)^getBits(i, 38));
    }
    break;
// Expression #46, Amount of variables: 42
case 46:
    for (long i = element; i < (((long) 1) << 42); i += maxCores) {
        result = result && (getBits(i, 0)&&!getBits(i, 1)||getBits(i, 2)^!getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)^getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)&&getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)&&!getBits(i, 13)||getBits(i, 15)||!getBits(i, 16)||getBits(i, 17)&&!getBits(i, 18)||getBits(i, 19)||getBits(i, 20)||getBits(i, 21)||getBits(i, 22)||getBits(i, 23)&&getBits(i, 24)||getBits(i, 25)&&getBits(i, 26)||getBits(i, 27)||getBits(i, 28)||getBits(i, 6)&&getBits(i, 29)||getBits(i, 30)^getBits(i, 31)||getBits(i, 32)||getBits(i, 2)||getBits(i, 24)^!getBits(i, 33)||getBits(i, 34)||getBits(i, 6)||getBits(i, 35)^getBits(i, 36)||getBits(i, 37)||getBits(i, 38)||getBits(i, 39)&&getBits(i, 29)||getBits(i, 1)^!getBits(i, 20)||getBits(i, 40)&&getBits(i, 41));
    }
    break;
// Expression #47, Amount of variables: 43
case 47:
    for (long i = element; i < (((long) 1) << 43); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)^!getBits(i, 3)||getBits(i, 4)&&!getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)^getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)^getBits(i, 15)||getBits(i, 16)||getBits(i, 17)||getBits(i, 18)^!getBits(i, 19)||getBits(i, 20)||!getBits(i, 1)||getBits(i, 21)^getBits(i, 22)||getBits(i, 23)^!getBits(i, 24)||getBits(i, 25)||getBits(i, 26)||getBits(i, 27)&&getBits(i, 28)||getBits(i, 28)||getBits(i, 29)||getBits(i, 2)^getBits(i, 5)||getBits(i, 30)^!getBits(i, 29)||getBits(i, 31)||!getBits(i, 32)||getBits(i, 33)&&getBits(i, 34)||getBits(i, 35)||getBits(i, 2)||getBits(i, 5)||getBits(i, 5)||getBits(i, 36)||getBits(i, 37)||getBits(i, 38)||!getBits(i, 39)||getBits(i, 40)||getBits(i, 41)||getBits(i, 42)&&getBits(i, 29));
    }
    break;
// Expression #48, Amount of variables: 43
case 48:
    for (long i = element; i < (((long) 1) << 43); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)&&!getBits(i, 3)||getBits(i, 4)^getBits(i, 5)||getBits(i, 5)^!getBits(i, 6)||getBits(i, 7)^getBits(i, 8)||getBits(i, 9)^!getBits(i, 10)||getBits(i, 11)||getBits(i, 9)||getBits(i, 12)&&getBits(i, 13)||getBits(i, 14)&&!getBits(i, 15)||getBits(i, 16)&&getBits(i, 17)||getBits(i, 18)^getBits(i, 9)||getBits(i, 19)&&!getBits(i, 20)||getBits(i, 21)&&getBits(i, 22)||getBits(i, 23)&&getBits(i, 24)||getBits(i, 25)||!getBits(i, 26)||getBits(i, 27)&&getBits(i, 28)||getBits(i, 14)||!getBits(i, 15)||getBits(i, 9)||getBits(i, 29)||getBits(i, 30)^getBits(i, 24)||getBits(i, 31)&&getBits(i, 19)||getBits(i, 32)&&getBits(i, 33)||getBits(i, 3)&&!getBits(i, 24)||getBits(i, 34)||getBits(i, 35)||getBits(i, 36)&&getBits(i, 26)||getBits(i, 37)&&getBits(i, 38)||getBits(i, 39)||!getBits(i, 40)||getBits(i, 41)&&!getBits(i, 42));
    }
    break;
// Expression #49, Amount of variables: 41
case 49:
    for (long i = element; i < (((long) 1) << 41); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&!getBits(i, 3)||getBits(i, 4)^!getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)&&getBits(i, 9)||getBits(i, 10)^!getBits(i, 11)||getBits(i, 12)&&getBits(i, 13)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)||getBits(i, 0)||getBits(i, 19)^getBits(i, 20)||getBits(i, 13)||getBits(i, 20)||getBits(i, 9)^getBits(i, 21)||getBits(i, 22)^getBits(i, 17)||getBits(i, 23)^getBits(i, 24)||getBits(i, 25)&&getBits(i, 26)||getBits(i, 27)&&getBits(i, 28)||getBits(i, 24)||getBits(i, 3)||getBits(i, 4)^getBits(i, 29)||getBits(i, 30)^getBits(i, 31)||getBits(i, 8)||getBits(i, 32)||getBits(i, 33)^getBits(i, 0)||getBits(i, 15)&&!getBits(i, 20)||getBits(i, 34)^getBits(i, 35)||getBits(i, 15)^getBits(i, 36)||getBits(i, 37)||getBits(i, 38)||getBits(i, 2)^!getBits(i, 39)||getBits(i, 15)&&getBits(i, 40));
    }
    break;
// Expression #50, Amount of variables: 40
case 50:
    for (long i = element; i < (((long) 1) << 40); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 0)||!getBits(i, 2)||getBits(i, 3)||getBits(i, 4)||getBits(i, 5)&&getBits(i, 6)||getBits(i, 7)&&getBits(i, 8)||getBits(i, 9)&&getBits(i, 10)||getBits(i, 11)&&!getBits(i, 12)||getBits(i, 13)||getBits(i, 2)||getBits(i, 14)||getBits(i, 1)||getBits(i, 0)||!getBits(i, 15)||getBits(i, 11)^getBits(i, 16)||getBits(i, 17)&&!getBits(i, 18)||getBits(i, 19)^getBits(i, 20)||getBits(i, 12)||getBits(i, 20)||getBits(i, 0)||getBits(i, 16)||getBits(i, 21)&&!getBits(i, 4)||getBits(i, 6)^getBits(i, 20)||getBits(i, 1)^getBits(i, 22)||getBits(i, 23)&&getBits(i, 4)||getBits(i, 4)^!getBits(i, 24)||getBits(i, 25)^getBits(i, 26)||getBits(i, 27)&&getBits(i, 28)||getBits(i, 29)&&getBits(i, 16)||getBits(i, 30)&&getBits(i, 31)||getBits(i, 32)||getBits(i, 33)||getBits(i, 22)||getBits(i, 34)||getBits(i, 35)||getBits(i, 36)||getBits(i, 37)^getBits(i, 17)||getBits(i, 38)&&getBits(i, 39));
    }
    break;
// Expression #51, Amount of variables: 51
case 51:
    for (long i = element; i < (((long) 1) << 51); i += maxCores) {
        result = result && (getBits(i, 0)^!getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 10)&&getBits(i, 11)||getBits(i, 12)&&!getBits(i, 13)||getBits(i, 14)^getBits(i, 15)||getBits(i, 16)^!getBits(i, 17)||getBits(i, 18)||getBits(i, 19)||getBits(i, 3)&&!getBits(i, 20)||getBits(i, 21)&&getBits(i, 22)||getBits(i, 23)&&getBits(i, 11)||getBits(i, 24)&&getBits(i, 25)||getBits(i, 26)||getBits(i, 27)||getBits(i, 28)||getBits(i, 29)||getBits(i, 12)||!getBits(i, 30)||getBits(i, 31)&&getBits(i, 5)||getBits(i, 32)||getBits(i, 33)||getBits(i, 34)&&getBits(i, 11)||getBits(i, 35)||getBits(i, 36)||getBits(i, 37)^getBits(i, 38)||getBits(i, 25)||getBits(i, 39)||getBits(i, 40)||getBits(i, 41)||getBits(i, 42)&&getBits(i, 43)||getBits(i, 44)^!getBits(i, 45)||getBits(i, 46)^getBits(i, 38)||getBits(i, 47)^getBits(i, 48)||getBits(i, 49)||getBits(i, 50)||getBits(i, 1)^!getBits(i, 50));
    }
    break;
// Expression #52, Amount of variables: 52
case 52:
    for (long i = element; i < (((long) 1) << 52); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)^getBits(i, 3)||getBits(i, 4)&&!getBits(i, 5)||getBits(i, 6)&&!getBits(i, 7)||getBits(i, 8)&&getBits(i, 9)||getBits(i, 10)&&getBits(i, 11)||getBits(i, 12)^getBits(i, 13)||getBits(i, 14)^getBits(i, 3)||getBits(i, 15)||getBits(i, 16)||getBits(i, 17)^getBits(i, 18)||getBits(i, 19)^getBits(i, 20)||getBits(i, 21)||getBits(i, 22)||getBits(i, 22)||getBits(i, 23)||getBits(i, 24)||!getBits(i, 25)||getBits(i, 26)||!getBits(i, 27)||getBits(i, 28)^getBits(i, 16)||getBits(i, 29)^!getBits(i, 30)||getBits(i, 31)||getBits(i, 32)||getBits(i, 33)||!getBits(i, 34)||getBits(i, 35)^getBits(i, 36)||getBits(i, 36)||!getBits(i, 37)||getBits(i, 38)&&getBits(i, 39)||getBits(i, 40)||getBits(i, 30)||getBits(i, 33)||getBits(i, 18)||getBits(i, 41)||getBits(i, 42)||getBits(i, 7)&&getBits(i, 43)||getBits(i, 44)^getBits(i, 45)||getBits(i, 46)&&getBits(i, 47)||getBits(i, 42)||getBits(i, 48)||getBits(i, 18)||getBits(i, 49)||getBits(i, 50)||getBits(i, 51));
    }
    break;
// Expression #53, Amount of variables: 54
case 53:
    for (long i = element; i < (((long) 1) << 54); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)^!getBits(i, 5)||getBits(i, 3)&&getBits(i, 6)||getBits(i, 7)&&getBits(i, 8)||getBits(i, 9)||getBits(i, 10)||getBits(i, 11)^getBits(i, 12)||getBits(i, 13)||getBits(i, 10)||getBits(i, 14)^getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)^getBits(i, 19)||getBits(i, 4)^getBits(i, 20)||getBits(i, 21)&&getBits(i, 20)||getBits(i, 22)^getBits(i, 23)||getBits(i, 24)||getBits(i, 25)||getBits(i, 26)^getBits(i, 27)||getBits(i, 28)||getBits(i, 17)||getBits(i, 29)^!getBits(i, 30)||getBits(i, 31)&&!getBits(i, 32)||getBits(i, 31)^getBits(i, 33)||getBits(i, 34)^getBits(i, 35)||getBits(i, 36)&&!getBits(i, 37)||getBits(i, 38)&&!getBits(i, 39)||getBits(i, 40)&&getBits(i, 41)||getBits(i, 42)^getBits(i, 43)||getBits(i, 44)||!getBits(i, 45)||getBits(i, 8)||getBits(i, 46)||getBits(i, 47)^!getBits(i, 48)||getBits(i, 47)&&getBits(i, 49)||getBits(i, 43)&&getBits(i, 50)||getBits(i, 51)^getBits(i, 52)||getBits(i, 15)&&getBits(i, 53));
    }
    break;
// Expression #54, Amount of variables: 47
case 54:
    for (long i = element; i < (((long) 1) << 47); i += maxCores) {
        result = result && (getBits(i, 0)^getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)||!getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 10)&&!getBits(i, 11)||getBits(i, 12)^getBits(i, 13)||getBits(i, 14)&&getBits(i, 15)||getBits(i, 16)||!getBits(i, 17)||getBits(i, 18)&&getBits(i, 19)||getBits(i, 20)&&getBits(i, 21)||getBits(i, 22)^getBits(i, 23)||getBits(i, 24)||!getBits(i, 15)||getBits(i, 19)&&getBits(i, 25)||getBits(i, 26)^getBits(i, 27)||getBits(i, 28)&&getBits(i, 24)||getBits(i, 29)&&getBits(i, 30)||getBits(i, 7)&&getBits(i, 17)||getBits(i, 31)||getBits(i, 30)||getBits(i, 32)^!getBits(i, 33)||getBits(i, 34)||getBits(i, 35)||getBits(i, 15)||!getBits(i, 36)||getBits(i, 15)||!getBits(i, 37)||getBits(i, 38)&&!getBits(i, 38)||getBits(i, 39)&&getBits(i, 7)||getBits(i, 5)^getBits(i, 40)||getBits(i, 24)^getBits(i, 41)||getBits(i, 42)^getBits(i, 24)||getBits(i, 32)&&getBits(i, 19)||getBits(i, 43)^getBits(i, 44)||getBits(i, 3)&&!getBits(i, 10)||getBits(i, 44)||!getBits(i, 45)||getBits(i, 23)^getBits(i, 46));
    }
    break;
// Expression #55, Amount of variables: 55
case 55:
    for (long i = element; i < (((long) 1) << 55); i += maxCores) {
        result = result && (getBits(i, 0)||getBits(i, 1)||getBits(i, 2)||!getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)^getBits(i, 7)||getBits(i, 3)^!getBits(i, 8)||getBits(i, 9)||getBits(i, 10)||getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)||getBits(i, 15)&&getBits(i, 16)||getBits(i, 17)^getBits(i, 18)||getBits(i, 19)^getBits(i, 20)||getBits(i, 21)&&getBits(i, 22)||getBits(i, 22)&&getBits(i, 23)||getBits(i, 24)^getBits(i, 25)||getBits(i, 26)^getBits(i, 27)||getBits(i, 5)||getBits(i, 28)||getBits(i, 7)||getBits(i, 29)||getBits(i, 30)^getBits(i, 31)||getBits(i, 32)&&getBits(i, 26)||getBits(i, 33)^getBits(i, 34)||getBits(i, 35)||getBits(i, 36)||getBits(i, 37)^getBits(i, 38)||getBits(i, 39)^getBits(i, 40)||getBits(i, 41)&&getBits(i, 42)||getBits(i, 43)||!getBits(i, 44)||getBits(i, 45)||getBits(i, 46)||getBits(i, 47)^getBits(i, 21)||getBits(i, 2)^getBits(i, 48)||getBits(i, 49)||getBits(i, 50)||getBits(i, 51)^getBits(i, 23)||getBits(i, 48)&&!getBits(i, 44)||getBits(i, 52)&&getBits(i, 53)||getBits(i, 54)||getBits(i, 8)||getBits(i, 21)||getBits(i, 25));
    }
    break;
// Expression #56, Amount of variables: 60
case 56:
    for (long i = element; i < (((long) 1) << 60); i += maxCores) {
        result = result && (getBits(i, 0)&&!getBits(i, 0)||getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||!getBits(i, 6)||getBits(i, 7)||getBits(i, 8)||getBits(i, 9)^getBits(i, 10)||getBits(i, 11)&&getBits(i, 12)||getBits(i, 7)^!getBits(i, 13)||getBits(i, 14)||getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)&&getBits(i, 19)||getBits(i, 20)^getBits(i, 21)||getBits(i, 22)||!getBits(i, 23)||getBits(i, 24)||getBits(i, 22)||getBits(i, 25)^!getBits(i, 26)||getBits(i, 27)&&getBits(i, 28)||getBits(i, 29)^!getBits(i, 30)||getBits(i, 31)||getBits(i, 32)||getBits(i, 33)||getBits(i, 34)||getBits(i, 35)&&getBits(i, 36)||getBits(i, 37)&&getBits(i, 38)||getBits(i, 39)^getBits(i, 21)||getBits(i, 40)^getBits(i, 41)||getBits(i, 42)^getBits(i, 43)||getBits(i, 19)||!getBits(i, 44)||getBits(i, 45)&&getBits(i, 46)||getBits(i, 47)&&getBits(i, 48)||getBits(i, 49)||getBits(i, 43)||getBits(i, 0)^getBits(i, 50)||getBits(i, 51)^getBits(i, 52)||getBits(i, 53)&&getBits(i, 54)||getBits(i, 50)^getBits(i, 12)||getBits(i, 55)&&getBits(i, 56)||getBits(i, 57)&&getBits(i, 58)||getBits(i, 44)&&getBits(i, 59));
    }
    break;
// Expression #57, Amount of variables: 57
case 57:
    for (long i = element; i < (((long) 1) << 57); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)&&getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 7)||getBits(i, 10)||getBits(i, 11)^getBits(i, 12)||getBits(i, 9)&&getBits(i, 13)||getBits(i, 14)^getBits(i, 15)||getBits(i, 16)^!getBits(i, 17)||getBits(i, 18)&&getBits(i, 19)||getBits(i, 20)||!getBits(i, 21)||getBits(i, 22)^getBits(i, 23)||getBits(i, 24)^getBits(i, 25)||getBits(i, 3)||getBits(i, 16)||getBits(i, 26)^getBits(i, 27)||getBits(i, 28)||getBits(i, 16)||getBits(i, 15)^!getBits(i, 12)||getBits(i, 29)||getBits(i, 12)||getBits(i, 30)&&getBits(i, 31)||getBits(i, 32)&&!getBits(i, 33)||getBits(i, 34)^!getBits(i, 31)||getBits(i, 35)&&getBits(i, 12)||getBits(i, 25)&&!getBits(i, 36)||getBits(i, 37)||getBits(i, 12)||getBits(i, 38)||getBits(i, 39)||getBits(i, 40)&&getBits(i, 15)||getBits(i, 41)||!getBits(i, 42)||getBits(i, 43)||getBits(i, 44)||getBits(i, 12)||getBits(i, 45)||getBits(i, 42)&&getBits(i, 46)||getBits(i, 47)&&getBits(i, 48)||getBits(i, 49)||getBits(i, 50)||getBits(i, 51)||!getBits(i, 52)||getBits(i, 53)||getBits(i, 54)||getBits(i, 55)||getBits(i, 56));
    }
    break;
// Expression #58, Amount of variables: 58
case 58:
    for (long i = element; i < (((long) 1) << 58); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)&&getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)||getBits(i, 7)||getBits(i, 8)||getBits(i, 9)||getBits(i, 10)^getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)||!getBits(i, 15)||getBits(i, 16)^getBits(i, 17)||getBits(i, 18)^getBits(i, 19)||getBits(i, 20)||getBits(i, 21)||getBits(i, 22)||getBits(i, 23)||getBits(i, 24)||!getBits(i, 25)||getBits(i, 26)&&getBits(i, 27)||getBits(i, 28)&&getBits(i, 29)||getBits(i, 30)||!getBits(i, 31)||getBits(i, 32)^getBits(i, 33)||getBits(i, 34)||getBits(i, 35)||getBits(i, 36)&&getBits(i, 36)||getBits(i, 11)||getBits(i, 7)||getBits(i, 31)||getBits(i, 37)||getBits(i, 38)||!getBits(i, 35)||getBits(i, 39)&&!getBits(i, 40)||getBits(i, 41)||getBits(i, 37)||getBits(i, 6)||getBits(i, 42)||getBits(i, 31)&&!getBits(i, 36)||getBits(i, 43)||getBits(i, 37)||getBits(i, 44)||getBits(i, 45)||getBits(i, 46)&&getBits(i, 47)||getBits(i, 30)&&!getBits(i, 39)||getBits(i, 48)&&!getBits(i, 36)||getBits(i, 49)||getBits(i, 30)||getBits(i, 35)&&getBits(i, 50)||getBits(i, 51)||getBits(i, 52)||getBits(i, 53)&&getBits(i, 54)||getBits(i, 48)^getBits(i, 55)||getBits(i, 56)||getBits(i, 57));
    }
    break;
// Expression #59, Amount of variables: 61
case 59:
    for (long i = element; i < (((long) 1) << 61); i += maxCores) {
        result = result && (getBits(i, 0)&&getBits(i, 1)||getBits(i, 2)||getBits(i, 3)||getBits(i, 4)||getBits(i, 5)||getBits(i, 6)&&getBits(i, 7)||getBits(i, 8)^getBits(i, 9)||getBits(i, 7)&&getBits(i, 10)||getBits(i, 11)||getBits(i, 12)||getBits(i, 13)||getBits(i, 14)||getBits(i, 15)&&getBits(i, 16)||getBits(i, 17)&&getBits(i, 18)||getBits(i, 19)^getBits(i, 20)||getBits(i, 21)&&getBits(i, 22)||getBits(i, 23)^getBits(i, 24)||getBits(i, 25)^getBits(i, 26)||getBits(i, 27)&&!getBits(i, 28)||getBits(i, 29)^getBits(i, 30)||getBits(i, 31)||getBits(i, 32)||getBits(i, 33)^getBits(i, 34)||getBits(i, 35)||getBits(i, 36)||getBits(i, 14)||!getBits(i, 37)||getBits(i, 38)||!getBits(i, 39)||getBits(i, 40)^getBits(i, 41)||getBits(i, 38)&&getBits(i, 28)||getBits(i, 42)^getBits(i, 43)||getBits(i, 12)||!getBits(i, 12)||getBits(i, 44)&&getBits(i, 45)||getBits(i, 20)^getBits(i, 46)||getBits(i, 45)&&!getBits(i, 47)||getBits(i, 48)&&getBits(i, 49)||getBits(i, 26)&&getBits(i, 50)||getBits(i, 51)&&getBits(i, 52)||getBits(i, 53)||getBits(i, 54)||getBits(i, 26)||getBits(i, 55)||getBits(i, 56)||getBits(i, 43)||getBits(i, 57)&&getBits(i, 58)||getBits(i, 30)&&getBits(i, 26)||getBits(i, 49)||!getBits(i, 10)||getBits(i, 59)&&getBits(i, 60));
    }
    break;
  }
}
