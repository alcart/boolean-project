#include <sys/time.h>
#include <stdio.h>
#include <signal.h> 
#include "gpu_boolean.cu"

struct timeval start, end;
FILE* data;

long max_iterations(int index);
void starttime() {
  gettimeofday( &start, 0);
}

double endtime() {
  gettimeofday( &end, 0);
  double elapsed = ( end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0;
  return elapsed;
}

void handle_sigint(int sig) {
  if (sig == SIGTERM || sig == SIGINT) {
    if (data) fclose(data);
  }
  signal (sig, SIG_DFL);
  raise (sig); 
}

void test() {
  data = fopen("./data.csv", "w");
  //Save data to file
  fputs("Operation,Number of iterations,Number of cores,Runtime\n", data);
  //Output data to console
  printf("Operation\tNumber of iterations\tNumber of cores\tRuntime\n"); 
  
  //Run through all expressions
  for (int i = 1; i <= 59; i++) {
    int numThreads = 1024;
    long maxIterations = max_iterations(i);

    for (int k = 4; k >= 0; k--) {
      int total_runs = 5;
      long numCores = maxIterations/(numThreads*k + 1);
      if (numCores == 0) numCores = 1;
      
      if (numCores >= ((long)2 << 41))
         numCores = (long)2 << 41;
      
      //Save data to file
      fprintf(data, "%d,%ld,%ld,", i, maxIterations, numCores); 
      //Output data to console
      printf("%d\t%ld\t%ld\t", i, maxIterations, numCores); 
      double elapsed_total_tests = 0; 
      // Run 5 tests 
      for (int j = 0; j < total_runs; j++) {
        starttime();
        //Cuda code
        char* gpuResult;
        char result = 1;
        hipMalloc(&gpuResult, sizeof(char));
        hipMemcpy(gpuResult, &result, sizeof(char), hipMemcpyHostToDevice);
        gpu_boolean_matcher<<<numCores, numThreads>>>(gpuResult, i);
        hipMemcpy(&result, gpuResult, sizeof(char), hipMemcpyDeviceToHost);
        hipFree(&gpuResult);
        elapsed_total_tests += endtime();
      }
      
      double average_runtime = elapsed_total_tests/total_runs;
     
      //Save data to file
      fprintf(data, "%f ms\n", average_runtime);
      //Output data to console
      printf("%f ms\n", average_runtime);
      fflush(stdout);
      fflush(data);
    }
    //Print total_runtime with cores
  } 
  fclose(data);
}

long max_iterations(int index) {
  
    switch (index) {
	// Expression #1, Amount of variables: 2
	case 1:
	    return (((long) 1) << 2);
	// Expression #2, Amount of variables: 2
	case 2:
	    return (((long) 1) << 2);
	// Expression #3, Amount of variables: 2
	case 3:
	    return (((long) 1) << 2);
	// Expression #4, Amount of variables: 3
	case 4:
	    return (((long) 1) << 3);
	// Expression #5, Amount of variables: 4
	case 5:
	    return (((long) 1) << 4);
	// Expression #6, Amount of variables: 5
	case 6:
	    return (((long) 1) << 5);
	// Expression #7, Amount of variables: 20
	case 7:
	    return (((long) 1) << 20);
	// Expression #8, Amount of variables: 21
	case 8:
	    return (((long) 1) << 21);
	// Expression #9, Amount of variables: 25
	case 9:
	    return (((long) 1) << 25);
	// Expression #10, Amount of variables: 21
	case 10:
	    return (((long) 1) << 21);
	// Expression #11, Amount of variables: 30
	case 11:
	    return (((long) 1) << 30);
	// Expression #12, Amount of variables: 21
	case 12:
	    return (((long) 1) << 21);
	// Expression #13, Amount of variables: 39
	case 13:
	    return (((long) 1) << 39);
	// Expression #14, Amount of variables: 45
	case 14:
	    return (((long) 1) << 45);
	// Expression #15, Amount of variables: 50
	case 15:
	    return (((long) 1) << 50);
	// Expression #16, Amount of variables: 60
	case 16:
	    return (((long) 1) << 60);
	// Expression #17, Amount of variables: 61
	case 17:
	    return (((long) 1) << 61);
	// Expression #18, Amount of variables: 60
	case 18:
	    return (((long) 1) << 60);
	// Expression #19, Amount of variables: 62
	case 19:
	    return (((long) 1) << 62);
	// Expression #20, Amount of variables: 60
	case 20:
	    return (((long) 1) << 60);
	// Expression #21, Amount of variables: 63
	case 21:
	    return (((long) 1) << 63);
	// Expression #22, Amount of variables: 2
	case 22:
	    return (((long) 1) << 2);
	// Expression #23, Amount of variables: 4
	case 23:
	    return (((long) 1) << 4);
	// Expression #24, Amount of variables: 5
	case 24:
	    return (((long) 1) << 5);
	// Expression #25, Amount of variables: 8
	case 25:
	    return (((long) 1) << 8);
	// Expression #26, Amount of variables: 9
	case 26:
	    return (((long) 1) << 9);
	// Expression #27, Amount of variables: 12
	case 27:
	    return (((long) 1) << 12);
	// Expression #28, Amount of variables: 10
	case 28:
	    return (((long) 1) << 10);
	// Expression #29, Amount of variables: 12
	case 29:
	    return (((long) 1) << 12);
	// Expression #30, Amount of variables: 16
	case 30:
	    return (((long) 1) << 16);
	// Expression #31, Amount of variables: 18
	case 31:
	    return (((long) 1) << 18);
	// Expression #32, Amount of variables: 18
	case 32:
	    return (((long) 1) << 18);
	// Expression #33, Amount of variables: 21
	case 33:
	    return (((long) 1) << 21);
	// Expression #34, Amount of variables: 24
	case 34:
	    return (((long) 1) << 24);
	// Expression #35, Amount of variables: 26
	case 35:
	    return (((long) 1) << 26);
	// Expression #36, Amount of variables: 28
	case 36:
	    return (((long) 1) << 28);
	// Expression #37, Amount of variables: 29
	case 37:
	    return (((long) 1) << 29);
	// Expression #38, Amount of variables: 27
	case 38:
	    return (((long) 1) << 27);
	// Expression #39, Amount of variables: 29
	case 39:
	    return (((long) 1) << 29);
	// Expression #40, Amount of variables: 31
	case 40:
	    return (((long) 1) << 31);
	// Expression #41, Amount of variables: 34
	case 41:
	    return (((long) 1) << 34);
	// Expression #42, Amount of variables: 35
	case 42:
	    return (((long) 1) << 35);
	// Expression #43, Amount of variables: 36
	case 43:
	    return (((long) 1) << 36);
	// Expression #44, Amount of variables: 40
	case 44:
	    return (((long) 1) << 40);
	// Expression #45, Amount of variables: 39
	case 45:
	    return (((long) 1) << 39);
	// Expression #46, Amount of variables: 42
	case 46:
	    return (((long) 1) << 42);
	// Expression #47, Amount of variables: 43
	case 47:
	    return (((long) 1) << 43);
	// Expression #48, Amount of variables: 43
	case 48:
	    return (((long) 1) << 43);
	// Expression #49, Amount of variables: 41
	case 49:
	    return (((long) 1) << 41);
	// Expression #50, Amount of variables: 40
	case 50:
	    return (((long) 1) << 40);
	// Expression #51, Amount of variables: 51
	case 51:
	    return (((long) 1) << 51);
	// Expression #52, Amount of variables: 52
	case 52:
	    return (((long) 1) << 52);
	// Expression #53, Amount of variables: 54
	case 53:
	    return (((long) 1) << 54);
	// Expression #54, Amount of variables: 47
	case 54:
	    return (((long) 1) << 47);
	// Expression #55, Amount of variables: 55
	case 55:
	    return (((long) 1) << 55);
	// Expression #56, Amount of variables: 60
	case 56:
	    return (((long) 1) << 60);
	// Expression #57, Amount of variables: 57
	case 57:
	    return (((long) 1) << 57);
	// Expression #58, Amount of variables: 58
	case 58:
	    return (((long) 1) << 58);
	// Expression #59, Amount of variables: 61
	case 59:
	    return (((long) 1) << 61);	
      }
      return 0; // Should not happen
}

int main() {
 signal(SIGINT, handle_sigint); 
 test();
 return 0;
}
