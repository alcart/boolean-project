#include <sys/time.h>
#include <stdio.h>
#include <signal.h> 
#include "gpu_boolean.cu"

struct timeval start, end;
FILE* data;

unsigned long max_iterations(int index);
void starttime() {
  gettimeofday( &start, 0);
}

double endtime() {
  gettimeofday( &end, 0);
  double elapsed = ( end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0;
  return elapsed;
}

// function handler for SIGNALS to avoid loss of data
void handle_sigint(int sig) {
  if (sig == SIGTERM || sig == SIGINT) {
    if (data) fclose(data);
  }
  signal (sig, SIG_DFL);
  raise (sig); 
}

void test() {
  data = fopen("./data.csv", "w");
  //Save data to file
  fputs("Operation,Number of iterations,Number of cores,Runtime\n", data);
  //Output data to console
  printf("Operation\tNumber of iterations\tNumber of cores\tRuntime\n"); 
  
  //Run through all expressions
  for (int i = 0; i <= 59; i++) {
    int numThreads = 1024;
    int maxNumberOfCores;

    if (i == 16) i = 22;

    unsigned long maxIterations = max_iterations(i);

    for (int j = 4; j > 0; j--) {
        unsigned long numCores = maxIterations/(numThreads*j);
        if (numCores == 0) numCores = 1;
        
        if (numCores >= (((unsigned long)1 << 31) - 1))
           numCores = (((unsigned long)1 << 31) - 1);
        
        //Save data to file
        fprintf(data, "%d,%lu,%lu,", i, maxIterations, numCores); 
        //Output data to console
        printf("%d\t%lu\t%lu\t", i, maxIterations, numCores); 
         
        starttime();
        //Cuda code
        char* gpuResult;
        char result = 1;
        hipMalloc(&gpuResult, sizeof(char));
        hipMemcpy(gpuResult, &result, sizeof(char), hipMemcpyHostToDevice);
        gpu_boolean_matcher<<<numCores, numThreads>>>(gpuResult, i);
        hipMemcpy(&result, gpuResult, sizeof(char), hipMemcpyDeviceToHost);
        hipFree(&gpuResult);
        double runtime = endtime();

        //Save data to file
        fprintf(data, "%f ms\n", runtime);
        //Output data to console
        printf("%f ms\n", runtime);
        fflush(stdout);
        fflush(data);
        //Print total_runtime with cores
 
    }
  } 
  fclose(data);
}

unsigned long max_iterations(int index) {
  
    switch (index) {
	// Expression #1, Amount of variables: 2
	case 1:
	    return (((unsigned long) 1) << 2);
	// Expression #2, Amount of variables: 2
	case 2:
	    return (((unsigned long) 1) << 2);
	// Expression #3, Amount of variables: 2
	case 3:
	    return (((unsigned long) 1) << 2);
	// Expression #4, Amount of variables: 3
	case 4:
	    return (((unsigned long) 1) << 3);
	// Expression #5, Amount of variables: 4
	case 5:
	    return (((unsigned long) 1) << 4);
	// Expression #6, Amount of variables: 5
	case 6:
	    return (((unsigned long) 1) << 5);
	// Expression #7, Amount of variables: 20
	case 7:
	    return (((unsigned long) 1) << 20);
	// Expression #8, Amount of variables: 21
	case 8:
	    return (((unsigned long) 1) << 21);
	// Expression #9, Amount of variables: 25
	case 9:
	    return (((unsigned long) 1) << 25);
	// Expression #10, Amount of variables: 21
	case 10:
	    return (((unsigned long) 1) << 21);
	// Expression #11, Amount of variables: 30
	case 11:
	    return (((unsigned long) 1) << 30);
	// Expression #12, Amount of variables: 21
	case 12:
	    return (((unsigned long) 1) << 21);
	// Expression #13, Amount of variables: 39
	case 13:
	    return (((unsigned long) 1) << 39);
	// Expression #14, Amount of variables: 45
	case 14:
	    return (((unsigned long) 1) << 45);
	// Expression #15, Amount of variables: 50
	case 15:
	    return (((unsigned long) 1) << 50);
	// Expression #16, Amount of variables: 60
	case 16:
	    return (((unsigned long) 1) << 60);
	// Expression #17, Amount of variables: 61
	case 17:
	    return (((unsigned long) 1) << 61);
	// Expression #18, Amount of variables: 60
	case 18:
	    return (((unsigned long) 1) << 60);
	// Expression #19, Amount of variables: 62
	case 19:
	    return (((unsigned long) 1) << 62);
	// Expression #20, Amount of variables: 60
	case 20:
	    return (((unsigned long) 1) << 60);
	// Expression #21, Amount of variables: 63
	case 21:
	    return (((unsigned long) 1) << 63);
	// Expression #22, Amount of variables: 2
	case 22:
	    return (((unsigned long) 1) << 2);
	// Expression #23, Amount of variables: 4
	case 23:
	    return (((unsigned long) 1) << 4);
	// Expression #24, Amount of variables: 5
	case 24:
	    return (((unsigned long) 1) << 5);
	// Expression #25, Amount of variables: 8
	case 25:
	    return (((unsigned long) 1) << 8);
	// Expression #26, Amount of variables: 9
	case 26:
	    return (((unsigned long) 1) << 9);
	// Expression #27, Amount of variables: 12
	case 27:
	    return (((unsigned long) 1) << 12);
	// Expression #28, Amount of variables: 10
	case 28:
	    return (((unsigned long) 1) << 10);
	// Expression #29, Amount of variables: 12
	case 29:
	    return (((unsigned long) 1) << 12);
	// Expression #30, Amount of variables: 16
	case 30:
	    return (((unsigned long) 1) << 16);
	// Expression #31, Amount of variables: 18
	case 31:
	    return (((unsigned long) 1) << 18);
	// Expression #32, Amount of variables: 18
	case 32:
	    return (((unsigned long) 1) << 18);
	// Expression #33, Amount of variables: 21
	case 33:
	    return (((unsigned long) 1) << 21);
	// Expression #34, Amount of variables: 24
	case 34:
	    return (((unsigned long) 1) << 24);
	// Expression #35, Amount of variables: 26
	case 35:
	    return (((unsigned long) 1) << 26);
	// Expression #36, Amount of variables: 28
	case 36:
	    return (((unsigned long) 1) << 28);
	// Expression #37, Amount of variables: 29
	case 37:
	    return (((unsigned long) 1) << 29);
	// Expression #38, Amount of variables: 27
	case 38:
	    return (((unsigned long) 1) << 27);
	// Expression #39, Amount of variables: 29
	case 39:
	    return (((unsigned long) 1) << 29);
	// Expression #40, Amount of variables: 31
	case 40:
	    return (((unsigned long) 1) << 31);
	// Expression #41, Amount of variables: 34
	case 41:
	    return (((unsigned long) 1) << 34);
	// Expression #42, Amount of variables: 35
	case 42:
	    return (((unsigned long) 1) << 35);
	// Expression #43, Amount of variables: 36
	case 43:
	    return (((unsigned long) 1) << 36);
	// Expression #44, Amount of variables: 40
	case 44:
	    return (((unsigned long) 1) << 40);
	// Expression #45, Amount of variables: 39
	case 45:
	    return (((unsigned long) 1) << 39);
	// Expression #46, Amount of variables: 42
	case 46:
	    return (((unsigned long) 1) << 42);
	// Expression #47, Amount of variables: 43
	case 47:
	    return (((unsigned long) 1) << 43);
	// Expression #48, Amount of variables: 43
	case 48:
	    return (((unsigned long) 1) << 43);
	// Expression #49, Amount of variables: 41
	case 49:
	    return (((unsigned long) 1) << 41);
	// Expression #50, Amount of variables: 40
	case 50:
	    return (((unsigned long) 1) << 40);
	// Expression #51, Amount of variables: 51
	case 51:
	    return (((unsigned long) 1) << 51);
	// Expression #52, Amount of variables: 52
	case 52:
	    return (((unsigned long) 1) << 52);
	// Expression #53, Amount of variables: 54
	case 53:
	    return (((unsigned long) 1) << 54);
	// Expression #54, Amount of variables: 47
	case 54:
	    return (((unsigned long) 1) << 47);
	// Expression #55, Amount of variables: 55
	case 55:
	    return (((unsigned long) 1) << 55);
	// Expression #56, Amount of variables: 60
	case 56:
	    return (((unsigned long) 1) << 60);
	// Expression #57, Amount of variables: 57
	case 57:
	    return (((unsigned long) 1) << 57);
	// Expression #58, Amount of variables: 58
	case 58:
	    return (((unsigned long) 1) << 58);
	// Expression #59, Amount of variables: 61
	case 59:
	    return (((unsigned long) 1) << 61);	
      }
      return 0; // Should not happen
}

int main() {
 signal(SIGINT, handle_sigint); 
 test();
 return 0;
}
